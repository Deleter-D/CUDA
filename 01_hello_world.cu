
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(int argc, char const *argv[])
{
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}