#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../utils/common.cuh"
#include "../utils/data.cuh"

/*
    使用如下命令分析代码
    nsys nvprof /path/to/out/02_pageable_memory
*/

int main(int argc, char const *argv[])
{
    setDevice();

    unsigned int size = 1 << 22;
    unsigned int bytes = size * sizeof(float);

    float *h_a = (float *)malloc(bytes);

    initializaData<float>(h_a, size);

    float *d_a;
    ERROR_CHECK(hipMalloc((void **)&d_a, bytes));

    ERROR_CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));

    ERROR_CHECK(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));

    ERROR_CHECK(hipFree(d_a));
    free(h_a);

    ERROR_CHECK(hipDeviceReset());

    return 0;
}
