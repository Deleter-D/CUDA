#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../utils/common.cuh"
#include "../utils/data.cuh"

/*
    使用如下命令分析全局存储效率和全局存储事务
    sudo ncu --target-processes all -k sumArraysWriteOffset --metrics smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct,l1tex__t_sectors_pipe_lsu_mem_global_op_st.sum /path/out/07_write_segment 0
    sudo ncu --target-processes all -k sumArraysWriteOffset --metrics smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct,l1tex__t_sectors_pipe_lsu_mem_global_op_st.sum /path/out/07_write_segment 11
    sudo ncu --target-processes all -k sumArraysWriteOffset --metrics smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct,l1tex__t_sectors_pipe_lsu_mem_global_op_st.sum /path/out/07_write_segment 128
*/

void sumArraysHost(float *A, float *B, float *C, const int size, int offset)
{
    for (int i = 0, j = offset; j < size; i++, j++)
        C[j] = A[i] + B[i];
}

__global__ void sumArraysWriteOffset(float *A, float *B, float *C, const int size, int offset)
{
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned j = tid + offset;
    if (j < size)
        C[j] = A[tid] + B[tid];
}

int main(int argc, char const *argv[])
{
    setDevice();

    int size = 1 << 22;
    printf("Array size: %d\n", size);
    size_t bytes = size * sizeof(float);

    int offset = 0;
    if (argc > 1)
        offset = atoi(argv[1]);

    dim3 block(512);
    dim3 grid((size + block.x - 1) / block.x);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(bytes);
    h_B = (float *)malloc(bytes);
    hostRef = (float *)malloc(bytes);
    gpuRef = (float *)malloc(bytes);

    initializeData<float>(h_A, size);
    initializeData<float>(h_B, size);

    sumArraysHost(h_A, h_B, hostRef, size, offset);

    float *d_A, *d_B, *d_C;
    ERROR_CHECK(hipMalloc((void **)&d_A, bytes));
    ERROR_CHECK(hipMalloc((void **)&d_B, bytes));
    ERROR_CHECK(hipMalloc((void **)&d_C, bytes));

    ERROR_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    float elapsedTime;
    ERROR_CHECK(hipEventCreate(&start));
    ERROR_CHECK(hipEventCreate(&stop));

    // 预热
    ERROR_CHECK(hipEventRecord(start));
    sumArraysWriteOffset<<<grid, block>>>(d_A, d_B, d_C, size, offset);
    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    ERROR_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));

    ERROR_CHECK(hipEventRecord(start));
    sumArraysWriteOffset<<<grid, block>>>(d_A, d_B, d_C, size, offset);
    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    ERROR_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("writeOffset<<<%d, %d>>>\toffset %4d\telapsed %f ms\n", grid.x, block.x, offset, elapsedTime);

    ERROR_CHECK(hipMemcpy(gpuRef, d_C, bytes, hipMemcpyDeviceToHost));
    ERROR_CHECK(hipDeviceSynchronize());

    checkResult<float>(hostRef, gpuRef, size);

    ERROR_CHECK(hipFree(d_A));
    ERROR_CHECK(hipFree(d_B));
    ERROR_CHECK(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    ERROR_CHECK(hipDeviceReset());

    return 0;
}