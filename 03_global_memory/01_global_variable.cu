#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../utils/common.cuh"

__device__ float devData;

__global__ void checkGlobalVariable()
{
    printf("Device:\tthe value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main(int argc, char const *argv[])
{
    setDevice();

    float value = 3.14f;
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host:\tcopied %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    ERROR_CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:\tthe value changed by the kernel to %f\n", value);

    ERROR_CHECK(hipDeviceReset());
    return 0;
}
