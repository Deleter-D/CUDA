#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../utils/common.cuh"

__device__ float devData;

__global__ void checkGlobalVariable()
{
    printf("Device:\tthe value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main(int argc, char const *argv[])
{
    setDevice();

    float value = 3.14f;

    // 使用hipMemcpyToSymbol的方式
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host:\tcopied %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    ERROR_CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:\tthe value changed by the kernel to %f\n", value);

    // 使用hipMemcpy的方式
    value = 3.14f;
    float *dptr;
    ERROR_CHECK(hipGetSymbolAddress((void **)&dptr, devData));
    ERROR_CHECK(hipMemcpy(dptr, &value, sizeof(float), hipMemcpyHostToDevice));
    printf("Host:\tcopied %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    ERROR_CHECK(hipMemcpy(&value, dptr, sizeof(float), hipMemcpyDeviceToHost));
    printf("Host:\tthe value changed by the kernel to %f\n", value);

    ERROR_CHECK(hipDeviceReset());
    return 0;
}
