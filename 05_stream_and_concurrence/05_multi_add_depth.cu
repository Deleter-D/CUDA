#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../utils/common.cuh"
#include "../utils/data.cuh"

void sumArraysOnHost(float *A, float *B, float *C, const int size)
{
    for (int idx = 0; idx < size; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float *A, float *B, float *C, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        for (int i = 0; i < 300; ++i) // 增加线程执行时间，便于在nsys中观察
        {
            C[idx] = A[idx] + B[idx];
        }
    }
}

int main(int argc, char const *argv[])
{
    int stream_count = 4;

    if (argc > 1) stream_count = atoi(argv[1]);

    // 通过环境变量调整流的行为
    char *env_name = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(env_name, "32", 1);
    printf("%s = %s\n", env_name, getenv(env_name));

    setDevice();

    hipDeviceProp_t prop;
    ERROR_CHECK(hipGetDeviceProperties(&prop, 0));
    if (prop.major < 3 || (prop.major == 3 && prop.minor < 5))
    {
        if (prop.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support Hyper-Q\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", prop.major, prop.minor, prop.multiProcessorCount);

    int size     = 1 << 18;
    size_t bytes = size * sizeof(float);
    printf("Array size: %d\n", size);

    float *h_A, *h_B, *hostRef, *gpuRef;
    ERROR_CHECK(hipHostAlloc((void **)&h_A, bytes, hipHostMallocDefault));
    ERROR_CHECK(hipHostAlloc((void **)&h_B, bytes, hipHostMallocDefault));
    ERROR_CHECK(hipHostAlloc((void **)&hostRef, bytes, hipHostMallocDefault));
    ERROR_CHECK(hipHostAlloc((void **)&gpuRef, bytes, hipHostMallocDefault));

    initializeData<float>(h_A, size);
    initializeData<float>(h_B, size);
    memset(hostRef, 0, bytes);
    memset(gpuRef, 0, bytes);

    sumArraysOnHost(h_A, h_B, hostRef, size);

    float *d_A, *d_B, *d_C;
    ERROR_CHECK(hipMalloc((void **)&d_A, bytes));
    ERROR_CHECK(hipMalloc((void **)&d_B, bytes));
    ERROR_CHECK(hipMalloc((void **)&d_C, bytes));

    hipEvent_t start, stop;
    ERROR_CHECK(hipEventCreate(&start));
    ERROR_CHECK(hipEventCreate(&stop));

    dim3 block(128);
    dim3 grid((size + block.x - 1) / block.x);

    ERROR_CHECK(hipEventRecord(start));
    ERROR_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));
    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    ERROR_CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    ERROR_CHECK(hipEventRecord(start));
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, size);
    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    float kernel_time;
    ERROR_CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    ERROR_CHECK(hipEventRecord(start));
    ERROR_CHECK(hipMemcpy(gpuRef, d_C, bytes, hipMemcpyDeviceToHost));
    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    ERROR_CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));

    float total_time = memcpy_h2d_time + kernel_time + memcpy_d2h_time;

    checkResult<float>(hostRef, gpuRef, size);

    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device:\t\t%f ms (%f GB/s)\n", memcpy_h2d_time, (bytes * 1e-6) / memcpy_h2d_time);
    printf(" Kernel execution:\t\t%f ms (%f GB/s)\n", kernel_time, (bytes * 1e-6) / kernel_time);
    printf(" Memcpy device to host:\t\t%f ms (%f GB/s)\n", memcpy_d2h_time, (2 * bytes * 1e-6) / memcpy_d2h_time);
    printf(" Total:\t\t\t\t%f ms (%f GB/s)\n", total_time, (2 * bytes * 1e-6) / total_time);

    // 网格级并行
    int size_per_stream     = size / stream_count;
    size_t bytes_per_stream = size_per_stream * sizeof(float);

    dim3 block2(128);
    dim3 grid2((size_per_stream + block2.x - 1) / block2.x);

    hipStream_t streams[stream_count];
    for (int i = 0; i < stream_count; i++)
    {
        ERROR_CHECK(hipStreamCreate(&streams[i]));
    }

    ERROR_CHECK(hipEventRecord(start));

    for (int i = 0; i < stream_count; i++)
    {
        int offset = i * size_per_stream;
        ERROR_CHECK(hipMemcpyAsync(&d_A[offset], &h_A[offset], bytes_per_stream, hipMemcpyHostToDevice, streams[i]));
        ERROR_CHECK(hipMemcpyAsync(&d_B[offset], &h_B[offset], bytes_per_stream, hipMemcpyHostToDevice, streams[i]));
        sumArrays<<<grid2, block2, 0, streams[i]>>>(&d_A[offset], &d_B[offset], &d_C[offset], size_per_stream);
        ERROR_CHECK(hipMemcpyAsync(&gpuRef[offset], &d_C[offset], bytes_per_stream, hipMemcpyDeviceToHost, streams[i]));
    }

    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));
    float stream_exec_time;
    ERROR_CHECK(hipEventElapsedTime(&stream_exec_time, start, stop));

    printf("Result from overlapping data transfers:\n");
    printf(" overlap with %d streams:\t%f ms (%f GB/s)\n", stream_count, stream_exec_time, (2 * bytes * 1e-6) / stream_exec_time);
    printf(" speedup:\t\t\t%f \n", (total_time - stream_exec_time) * 100.0f / total_time);

    ERROR_CHECK(hipGetLastError());

    checkResult<float>(hostRef, gpuRef, size);

    ERROR_CHECK(hipFree(d_A));
    ERROR_CHECK(hipFree(d_B));
    ERROR_CHECK(hipFree(d_C));

    ERROR_CHECK(hipHostFree(h_A));
    ERROR_CHECK(hipHostFree(h_B));
    ERROR_CHECK(hipHostFree(hostRef));
    ERROR_CHECK(hipHostFree(gpuRef));

    ERROR_CHECK(hipEventDestroy(start));
    ERROR_CHECK(hipEventDestroy(stop));

    for (int i = 0; i < stream_count; i++)
    {
        ERROR_CHECK(hipStreamDestroy(streams[i]));
    }

    ERROR_CHECK(hipDeviceReset());

    return 0;
}
