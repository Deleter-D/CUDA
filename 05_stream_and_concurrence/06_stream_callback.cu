#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "../utils/common.cuh"
#include "../utils/data.cuh"

#define SIZE (1 << 18)

void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void* data)
{
    printf("callback from stream %d\n", (int*)data);
}

void CUDART_CB my_callback2(void* data)
{
    printf("callback from stream %d\n", (int*)data);
}

__global__ void kernel_1(float* data)
{
    float sum = 0.0;
    for (int i = 0; i < SIZE; i++)
    {
        sum += sum + tan(0.1) * tan(0.1);
    }
    *data = sum;
}

__global__ void kernel_2(float* data)
{
    float sum = 0.0;
    for (int i = 0; i < SIZE; i++)
    {
        sum += sum + tan(0.1) * tan(0.1);
    }
    *data = sum;
}

__global__ void kernel_3(float* data)
{
    float sum = 0.0;
    for (int i = 0; i < SIZE; i++)
    {
        sum += sum + tan(0.1) * tan(0.1);
    }
    *data = sum;
}

__global__ void kernel_4(float* data)
{
    float sum = 0.0;
    for (int i = 0; i < SIZE; i++)
    {
        sum += sum + tan(0.1) * tan(0.1);
    }
    *data = sum;
}

int main(int argc, char const* argv[])
{
    int stream_count = 4;

    if (argc > 1) stream_count = atoi(argv[1]);

    // 通过环境变量调整流的行为
    char* env_name = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(env_name, "32", 1);
    printf("%s = %s\n", env_name, getenv(env_name));

    setDevice();

    hipDeviceProp_t prop;
    ERROR_CHECK(hipGetDeviceProperties(&prop, 0));
    if (prop.major < 3 || (prop.major == 3 && prop.minor < 5))
    {
        if (prop.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support Hyper-Q\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", prop.major, prop.minor, prop.multiProcessorCount);

    float* d_data;
    ERROR_CHECK(hipMalloc((void**)&d_data, sizeof(float)));

    hipStream_t* streams = (hipStream_t*)malloc(stream_count * sizeof(hipStream_t));

    for (int i = 0; i < stream_count; i++)
    {
        ERROR_CHECK(hipStreamCreate(&(streams[i])));
    }

    dim3 block(1);
    dim3 grid(1);

    float elapsedTime;
    hipEvent_t start, stop;
    ERROR_CHECK(hipEventCreate(&start));
    ERROR_CHECK(hipEventCreate(&stop));

    ERROR_CHECK(hipEventRecord(start));

    // 深度优先调度
    for (int i = 0; i < stream_count; i++)
    {
        kernel_1<<<grid, block, 0, streams[i]>>>(d_data);
        kernel_2<<<grid, block, 0, streams[i]>>>(d_data);
        kernel_3<<<grid, block, 0, streams[i]>>>(d_data);
        kernel_4<<<grid, block, 0, streams[i]>>>(d_data);

        ERROR_CHECK(hipStreamAddCallback(streams[i], my_callback, (void*)i, 0));
        ERROR_CHECK(hipLaunchHostFunc(streams[i], my_callback2, (void*)i));
    }

    ERROR_CHECK(hipEventRecord(stop));
    ERROR_CHECK(hipEventSynchronize(stop));

    ERROR_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Measured time for parallel execution = %f ms\n", elapsedTime);

    for (int i = 0; i < stream_count; i++)
    {
        ERROR_CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    ERROR_CHECK(hipEventDestroy(start));
    ERROR_CHECK(hipEventDestroy(stop));

    ERROR_CHECK(hipDeviceReset());

    return 0;
}
