
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char const *argv[])
{
    // >>> 主机内存分配及其初始化 <<<
    int elemCount     = 512;                       // 元素个数
    size_t bytesCount = elemCount * sizeof(float); // 总字节数
    float *h_A, *h_B, *h_C;
    // 申请主机内存
    h_A = (float *)malloc(bytesCount);
    h_B = (float *)malloc(bytesCount);
    h_C = (float *)malloc(bytesCount);
    // 初始化
    memset(h_A, 0, bytesCount);
    memset(h_B, 0, bytesCount);
    memset(h_C, 0, bytesCount);

    // >>> 设备内存分配及其初始化 <<<
    float *d_A, *d_B, *d_C;
    // 申请设备内存
    hipMalloc((void **)&d_A, bytesCount);
    hipMalloc((void **)&d_B, bytesCount);
    hipMalloc((void **)&d_C, bytesCount);
    // 初始化
    hipMemset(d_A, 0, bytesCount);
    hipMemset(d_B, 0, bytesCount);
    hipMemset(d_C, 0, bytesCount);

    // >>> 主机到设备的数据般移（将待计算数据般移到设备）<<<
    hipMemcpy(d_A, h_A, bytesCount, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytesCount, hipMemcpyHostToDevice);

    // 调用核函数...

    // >>> 设备到主机的数据般移（将计算结果般移到主机）<<<
    hipMemcpy(h_C, d_C, bytesCount, hipMemcpyDeviceToHost);

    // >>> 设备内存释放 <<<
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // >>> 主机内存释放 <<<
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}