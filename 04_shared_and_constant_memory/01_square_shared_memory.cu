#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../utils/common.cuh"
#include "../utils/data.cuh"

/*
    使用如下命令分析存储体冲突
    sudo ncu --target-processes all -k regex:"write" --metrics l1tex__data_pipe_lsu_wavefronts_mem_shared_op_ld.sum,l1tex__data_pipe_lsu_wavefronts_mem_shared_op_st.sum /path/to/01_square_shared_memory
*/

#define BDIMX 32
#define BDIMY 32

// 按行写入，按行读取
__global__ void writeRowReadRow(int *out)
{
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x];
}

// 按列写入，按列读取
__global__ void writeColReadCol(int *out)
{
    __shared__ int tile[BDIMX][BDIMY];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.x][threadIdx.y] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

// 按行写入，按列读取
__global__ void writeRowReadCol(int *out)
{
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

void printData(char *msg, int *in, const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

int main(int argc, char const *argv[])
{
    bool print = false;
    if (argc > 1)
        print = atoi(argv[1]);

    setDevice();

    hipSharedMemConfig config;
    ERROR_CHECK(hipDeviceGetSharedMemConfig(&config));
    printf("Bank Mode: %s\n", (config == hipSharedMemBankSizeFourByte ? "4-Byte" : "8-Byte"));

    int nx = BDIMX;
    int ny = BDIMY;

    size_t bytes = nx * ny * sizeof(int);

    int *gpuRef;
    gpuRef = (int *)malloc(bytes);

    int *d_C;
    ERROR_CHECK(hipMalloc((void **)&d_C, bytes));

    dim3 block(BDIMX, BDIMY);
    dim3 grid(1, 1);

    ERROR_CHECK(hipMemset(d_C, 0, bytes));
    writeRowReadRow<<<grid, block>>>(d_C);
    ERROR_CHECK(hipMemcpy(gpuRef, d_C, bytes, hipMemcpyDeviceToHost));
    if (print)
        printData("write row read row", gpuRef, nx * ny);

    ERROR_CHECK(hipMemset(d_C, 0, bytes));
    writeColReadCol<<<grid, block>>>(d_C);
    ERROR_CHECK(hipMemcpy(gpuRef, d_C, bytes, hipMemcpyDeviceToHost));
    if (print)
        printData("write col read col", gpuRef, nx * ny);

    ERROR_CHECK(hipMemset(d_C, 0, bytes));
    writeRowReadCol<<<grid, block>>>(d_C);
    ERROR_CHECK(hipMemcpy(gpuRef, d_C, bytes, hipMemcpyDeviceToHost));
    if (print)
        printData("write row read col", gpuRef, nx * ny);

    return 0;
}
